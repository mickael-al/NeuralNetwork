#include "hip/hip_runtime.h"
﻿
#include "kernel.h"
#include ""
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "CNNHelper.hpp"
#include <stdio.h>
#include "NeuralNetwork.hpp"

__global__ void InitNeuralNetwork(const NeuralSwapData * nld, float* weight_buffer)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nld->size)
    {
        return;
    }
    hiprandState state;
    hiprand_init(nld->seed, i, 0, &state);
    weight_buffer[i] = hiprand_uniform(&state) * 2.0f - 1.0f;
}

NeuralNetwork* createNeuralNetwork(NeuralNetworkData nnd)
{
    NeuralSwapData nld{};

    if (nnd.nb_input_layer <= 0 || nnd.nb_col_hiden_layer <= 0 || nnd.nb_hiden_layer <= 0 || nnd.nb_output_layer <= 0)
    {
        fprintf(stderr, "createNeuralNetwork failed! invalid input NeuralNetworkData\n");
        return nullptr;
    }

    nnd.activationSize = nnd.nb_input_layer + nnd.nb_col_hiden_layer * nnd.nb_hiden_layer + nnd.nb_output_layer;
    nnd.weightSize = nnd.nb_input_layer * nnd.nb_hiden_layer;
    for (int i = 0; i < nnd.nb_col_hiden_layer - 1; i++)
    {
        nnd.weightSize += nnd.nb_hiden_layer * nnd.nb_hiden_layer;
    }
    nnd.weightSize += nnd.nb_hiden_layer * nnd.nb_output_layer;
    int layerStep = 2 + nnd.nb_col_hiden_layer;
    nld.size = nnd.weightSize;
    float* weight_buffer = 0;
    float * activation_Buffer = 0;
    NeuralNetworkData* nnd_Buffer = 0;
    NeuralSwapData* nld_Buffer = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);

    cudaStatus = hipMalloc((void**)&weight_buffer, nnd.weightSize * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&activation_Buffer, nnd.activationSize * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&nnd_Buffer, sizeof(NeuralNetworkData));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&nld_Buffer, sizeof(NeuralSwapData));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(nnd_Buffer, &nnd, sizeof(NeuralNetworkData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(nld_Buffer, &nld, sizeof(NeuralSwapData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    dim3 dimGrid;
    dim3 dimBlock;
    
    CNNHelper::KernelDispath(nld.size, &deviceProp, &dimGrid, &dimBlock);
    InitNeuralNetwork<<<dimGrid, dimBlock>>>(nld_Buffer, weight_buffer);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "InitNeuralNetwork launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    return new NeuralNetwork(weight_buffer, activation_Buffer, nnd_Buffer, nld_Buffer);

Error:
    hipFree(weight_buffer);
    hipFree(activation_Buffer);
    hipFree(nnd_Buffer);
    hipFree(nld_Buffer);

    return nullptr;
}

void releaseNeuralNetwork(NeuralNetwork* network)
{
    delete network;
}

__global__ void addKernel(int* c, const int* a, const int* b,const int * size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size[0])
    {
        return;
    }
    c[i] = a[i] + b[i];
}

int addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    int* thread_size = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
  
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&thread_size, sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(thread_size, &size, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 dimGrid;
    dim3 dimBlock;

    CNNHelper::KernelDispath(size, &deviceProp, &dimGrid, &dimBlock);
    addKernel <<<dimGrid, dimBlock >>> (dev_c, dev_a, dev_b, thread_size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
