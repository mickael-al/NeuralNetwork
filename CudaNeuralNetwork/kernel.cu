#include "hip/hip_runtime.h"
﻿
#include "kernel.h"
#include ""
#include "hip/hip_runtime.h"
#include "CNNHelper.hpp"
#include <stdio.h>

int main()
{
    return 0;
}

NeuralNetwork* createNeuralNetwork()
{
    return nullptr;
}

void releaseNeuralNetwork(NeuralNetwork* network)
{

}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
  
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    int numBlocks;
    int blockSize;
    
    CNNHelper::KernelDispath(size, deviceProp.maxThreadsPerBlock, &numBlocks, &blockSize);    
    addKernel<<<numBlocks, blockSize >>>(dev_c, dev_a, dev_b);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
