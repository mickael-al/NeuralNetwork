#include "hip/hip_runtime.h"
﻿
#include "kernel.h"
#include ""
#include "hip/hip_runtime.h"
#include "CNNHelper.hpp"
#include <stdio.h>

int main()
{
    return 0;
}


__global__ void InitNeuralNetwork(const NeuralNetworkData * nnd,const NeuralSwapData * nld, float* weight_buffer)
{
    int i = threadIdx.x;
    if (i >= nld->size)
    {
        return;
    }
    weight_buffer[i] = 0.0f;
}

NeuralNetwork* createNeuralNetwork(NeuralNetworkData nnd)
{
    NeuralSwapData nld{};
    nnd.activationSize = nnd.nb_input_layer + nnd.nb_col_hiden_layer * nnd.nb_hiden_layer + nnd.nb_output_layer;
    nnd.weightSize = nnd.nb_input_layer * nnd.nb_hiden_layer;
    for (int i = 0; i < nnd.nb_col_hiden_layer - 1; i++)
    {
        nnd.weightSize += nnd.nb_hiden_layer * nnd.nb_hiden_layer;
    }
    nnd.weightSize += nnd.nb_hiden_layer * nnd.nb_output_layer;
    int layerStep = 2 + nnd.nb_col_hiden_layer;
    nld.size = nnd.weightSize;
    float* weight_buffer = 0;
    float * activation_Buffer = 0;
    NeuralNetworkData* nnd_Buffer = 0;
    NeuralSwapData* nld_Buffer = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&weight_buffer, nnd.weightSize * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&activation_Buffer, nnd.activationSize * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&nnd_Buffer, sizeof(NeuralNetworkData));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&nld_Buffer, sizeof(NeuralSwapData));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(nnd_Buffer, &nnd, sizeof(NeuralNetworkData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(nld_Buffer, &nld, sizeof(NeuralSwapData), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(weight_buffer);
    hipFree(activation_Buffer);
    hipFree(nnd_Buffer);
    hipFree(nld_Buffer);

    return nullptr;
}

void releaseNeuralNetwork(NeuralNetwork* network)
{

}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
  
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    int numBlocks;
    int blockSize;
    
    CNNHelper::KernelDispath(size, deviceProp.maxThreadsPerBlock, &numBlocks, &blockSize);    
    addKernel<<<numBlocks, blockSize >>>(dev_c, dev_a, dev_b);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
